#include "hip/hip_runtime.h"
#include <stdio.h>

__global__
void initWith(float num, float *a, int N)
{

  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride)
  {
    a[i] = num;
  }
}

__global__
void addVectorsInto(float *result, float *a, float *b, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride)
  {
    result[i] = a[i] + b[i];
  }
}

void checkElementsAre(float target, float *vector, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(vector[i] != target)
    {
      printf("FAIL: vector[%d] - %0.0f does not equal %0.0f\n", i, vector[i], target);
      exit(1);
    }
  }
  printf("Success! All values calculated correctly.\n");
}

int main()
{
  int deviceId;
  int numberOfSMs;

  hipGetDevice(&deviceId);
  hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
  printf("num SMs = %d\n", numberOfSMs);
  const int N = 2<<24;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;

  hipMallocManaged(&a, size);
  hipMallocManaged(&b, size);
  hipMallocManaged(&c, size);

  hipMemPrefetchAsync(a, size, deviceId);
  hipMemPrefetchAsync(b, size, deviceId);
  hipMemPrefetchAsync(c, size, deviceId);

  size_t threadsPerBlock;
  size_t numberOfBlocks;

  threadsPerBlock = 256;
  numberOfBlocks = 32 * numberOfSMs;

  hipError_t addVectorsErr;
  hipError_t asyncErr;

  initWith<<<numberOfBlocks, threadsPerBlock>>>(3, a, N);
  initWith<<<numberOfBlocks, threadsPerBlock>>>(4, b, N);
  initWith<<<numberOfBlocks, threadsPerBlock>>>(0, c, N);

  addVectorsInto<<<numberOfBlocks, threadsPerBlock>>>(c, a, b, N);

  addVectorsErr = hipGetLastError();
  if(addVectorsErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(addVectorsErr));

  asyncErr = hipDeviceSynchronize();
  if(asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));

  checkElementsAre(7, c, N);

  hipFree(a);
  hipFree(b);
  hipFree(c);
}

/*
1 prefetch
CUDA Memory Operation Statistics (by time):

 Time(%)  Total Time (ns)  Operations  Average  Minimum  Maximum              Operation            
 -------  ---------------  ----------  -------  -------  -------  ---------------------------------
   100.0         21203394         768  27608.6     1599   160828  [CUDA Unified Memory memcpy DtoH]

CUDA Kernel Statistics:

 Time(%)  Total Time (ns)  Instances   Average    Minimum  Maximum                      Name                    
 -------  ---------------  ---------  ----------  -------  --------  -------------------------------------------
    96.0         40732060          3  13577353.3   619599  20614807  initWith(float, float*, int)               
     4.0          1710960          1   1710960.0  1710960   1710960  addVectorsInto(float*, float*, float*, int)

2 prefetches
CUDA Kernel Statistics:

 Time(%)  Total Time (ns)  Instances   Average   Minimum  Maximum                      Name                    
 -------  ---------------  ---------  ---------  -------  --------  -------------------------------------------
    92.9         22514595          3  7504865.0   620718  21269383  initWith(float, float*, int)               
     7.1          1712015          1  1712015.0  1712015   1712015  addVectorsInto(float*, float*, float*, int)
                 40732060


CUDA Memory Operation Statistics (by time):

 Time(%)  Total Time (ns)  Operations  Average  Minimum  Maximum              Operation            
 -------  ---------------  ----------  -------  -------  -------  ---------------------------------
   100.0         21200406         768  27604.7     1599   160123  [CUDA Unified Memory memcpy DtoH]
                 21203394

3 prefetches
CUDA Kernel Statistics:

 Time(%)  Total Time (ns)  Instances   Average   Minimum  Maximum                     Name                    
 -------  ---------------  ---------  ---------  -------  -------  -------------------------------------------
    52.2          1867563          3   622521.0   618639   624558  initWith(float, float*, int)               
    47.8          1711696          1  1711696.0  1711696  1711696  addVectorsInto(float*, float*, float*, int)
                 40732060


CUDA Memory Operation Statistics (by time):

 Time(%)  Total Time (ns)  Operations  Average  Minimum  Maximum              Operation            
 -------  ---------------  ----------  -------  -------  -------  ---------------------------------
   100.0         21201498         768  27606.1     1599   160028  [CUDA Unified Memory memcpy DtoH]
*/